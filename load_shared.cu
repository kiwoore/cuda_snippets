#include "hip/hip_runtime.h"
__global__ void share_data(int *input)
{
	__shared__ int data[BLOCK_SIZE];
	
	data[threadIdx.x] = input[threadIdx.x];
	__syncthreads();
	
	// the state of the entire data array
	// is now well-defined for all threads
	// in this block
}

